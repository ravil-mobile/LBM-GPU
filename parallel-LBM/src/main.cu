#include "hip/hip_runtime.h"
//#ifdef GRAPHICS
//#define GLEW_STATIC
    #include<GL/glew.h>
    #include<GLFW/glfw3.h>
    #include<cuda_gl_interop.h>
//#endif


#include "hipblas.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <iostream>
#include <stdio.h>
#include <time.h>
#include <iostream>
#include <argp.h>
#include <vector>

#include "headers/parameters.h"
#include "headers/scanning.h"
#include "headers/boundary_conditions.h"
#include "headers/helper.h"
#include "headers/kernels.h"
#include "headers/domain.h"
#include "headers/hip/hip_runtime_api.h"

//#ifdef GRAPHICS
    #include "headers/graphics.h"
//#endif

int main(int argc, char **argv) {

    // read input arguments
    static struct argp_option options[] = {
        {"parameters",  'p', "FILE", 0, "Sets path to a parameter file"},
        {"boundary-cond", 'b', "FILE", 0, "Sets path to a boundary conditions file"},
        {"mesh", 'm', "FILE", 0, "Sets path to a mesh file"},
        {"threads-distr", 't', "FILE", 0, "Sets path to a threads distribution file"},
        { 0 }
    };

    struct Arguments arguments;
    arguments.parameters_file = NULL;
    arguments.boundary_cond_file = NULL;
    arguments.mesh_file = NULL;
    arguments.threads_distr_file = NULL;
    
    static char doc[] = "lbm -- 2D implementation of the Lattice Boltzmann Method in CUDA";
    static char args_doc[] = "--parameters=FILE --boundary-cond=FILE --mesh=FILE --threads-distr=FILE";

    static struct argp argp = {options, parse_opt, args_doc, doc};
    argp_parse (&argp, argc, argv, 0, 0, &arguments);

    ChooseGPU();
       
    // read input data
    struct SimulationParametes parameters;
    ReadParameterFile(arguments.parameters_file, parameters);

    struct Constants constants;
    constants.one = 3.0;
    constants.two = 4.5;
    constants.three = 1.5;

    struct BoundaryInfo boundary_info;
    ReadBoundaryFile(arguments.boundary_cond_file, boundary_info);

    // allocate constant data into the DEVICE
    CopyConstantsToDevice(parameters,
                          constants,
                          boundary_info,
                          coords,
                          weights);

#ifdef DEBUG    
    CheckConstMemoryCopy<<<1,1>>>();
    hipDeviceSynchronize();
#endif

    // allocate memory on the HOST
    int *flag_field = (int*)calloc(parameters.num_lattices, sizeof(int));
    ReadMeshFile(arguments.mesh_file, flag_field, parameters);

    // allocate and init DOMAIN on the DEVICE
    DomainHandler domain_handler;
    domain_handler.InitDomainOnDevice(parameters,
                                      flag_field);

    const Domain *domain = domain_handler.GetDeviceData();

    // allocate and init BOUNDARIES on the DEVICE
    BoundaryConditionsHandler bc_handler;
    
    ScanFlagField(flag_field,
                  domain_handler,
                  bc_handler,
                  parameters,
                  constants,
                  boundary_info);

    const BoundaryConditions *boundaries = bc_handler.GetDeviceData();

    CudaResourceDistr threads_distr;
    ReadThreadsDistrFile(arguments.threads_distr_file, threads_distr);

    CudaResourceDistr blocks_distr;
    ComputeBlcoksDistr(blocks_distr, threads_distr, parameters, boundaries); 
    
    // DEFAULT_THREAD_NUM - optimal thread distribution per block for small kernels
    const int DEFAULT_THREAD_NUM = 128; 
    int threads = 0;
    int blocks = 0;

#ifdef GRAPHICS
    // set OpenGL device
    glfwInit();

    // Create window 
    GLFWwindow* window = glfwCreateWindow(parameters.width,
                                          parameters.height,
                                          "Lattice Boltzmann",
                                          NULL,
                                          NULL);

    if (window == NULL) {
        std::cout << "ERROR: failed to create window." << std::endl; 
        glfwTerminate();
        exit(EXIT_FAILURE); 
    }

    // Bind object to context
    glfwMakeContextCurrent(window);

    if (glewInit() != GLEW_OK) {
        std::cout << "ERROR: failed to initialize glew" << std::endl; 
        exit(EXIT_FAILURE);
    }
    
    // first two paramters set location of left corner, other two are width and height	
    glViewport(0,0, parameters.width, parameters.height);
	
    // Register call back function with glfw
    glfwSetFramebufferSizeCallback( window, FramebufferSizeCallback);
    glfwSetMouseButtonCallback (window, MousePressCallback);
    glfwSetCursorPosCallback (window, CursorPosCallback); 
    // create buffer object to hold pixel data
    glGenBuffers(1, &buffer_object);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, buffer_object);
    glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, parameters.num_lattices * 4, NULL, GL_DYNAMIC_DRAW_ARB);

    // register buffer with cuda runtime
    hipGraphicsGLRegisterBuffer(&resource, buffer_object, cudaGraphicsMapFlagsNone);

    // uchar4 is defined by cuda
    uchar4* dev_ptr;
    size_t size; 
#endif

    // prepare cublas
    hipblasHandle_t handle;
    HANDLE_CUBLAS_ERROR(hipblasCreate(&handle));

    // prepare streams
    const int NUM_STREAMS = 6;
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; ++i)
        hipStreamCreate(&streams[i]);


    // for recording purposes
    //    getchar();
    // prepare timers
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // START of algorighm
    hipEventRecord(start, 0);
    for (int time = 0; (time < parameters.num_time_steps); ++time) {
        
        // perform streaming step
        threads = threads_distr.stream_device;
        blocks = blocks_distr.stream_device;
        StreamDevice<<<blocks, threads>>>(domain->dev_population,
                                          domain->dev_swap_buffer,
                                          domain->dev_flag_field);
        CUDA_CHECK_ERROR(); 
        
        // apply boundary consitions
        if (boundaries->num_wall_elements != 0) {
            threads = threads_distr.treat_non_slip_bc;
            blocks = blocks_distr.treat_non_slip_bc;
            TreatNonSlipBC<<<blocks, threads, 0,streams[0]>>>(boundaries->bc_wall_indices,
                                                              domain->dev_swap_buffer,
                                                              boundaries->num_wall_elements); 
            CUDA_CHECK_ERROR();
        }

        if (boundaries->num_moving_wall_elements != 0) {
            threads = threads_distr.treat_slip_bc;
            blocks = blocks_distr.treat_slip_bc;
            TreatSlipBC<<<blocks, threads, 0, streams[1]>>>(boundaries->bc_moving_wall_indices,
                                                            boundaries->bc_moving_wall_data,
                                                            domain->dev_density,
                                                             domain->dev_swap_buffer,
                                                             boundaries->num_moving_wall_elements);
            CUDA_CHECK_ERROR();
        }

        if (boundaries->num_inflow_elements != 0) {
            threads = threads_distr.treat_inflow_bc;
            blocks = blocks_distr.treat_inflow_bc;
            TreatInflowBC<<<blocks, threads, 0, streams[2]>>>(boundaries->bc_inflow_indices,
                                                              boundaries->bc_inflow_data,
                                                              domain->dev_density,
                                                              domain->dev_swap_buffer,
                                                              boundaries->num_inflow_elements);
            CUDA_CHECK_ERROR();
        }

        if (boundaries->num_outflow_elements != 0) {
            threads = threads_distr.treat_outflow_bc;
            blocks = blocks_distr.treat_outflow_bc;
            TreatOutflowBC<<<blocks, threads, 0,streams[3]>>>(boundaries->bc_outflow_indices,
                                                              domain->dev_velocity,
                                                              domain->dev_density,
                                                              domain->dev_swap_buffer,
                                                              boundaries->num_outflow_elements);
            CUDA_CHECK_ERROR();
        }


        HANDLE_ERROR(hipDeviceSynchronize());
        domain_handler.SwapPopulationFields(); 
       
        // perform collision step
        threads = threads_distr.update_density_field_device;
        blocks = blocks_distr.update_density_field_device;
        UpdateDensityFieldDevice<<<blocks, threads>>>(domain->dev_density,
                                                      domain->dev_population,
                                                      domain->dev_flag_field);

        CUDA_CHECK_ERROR(); 

        threads = threads_distr.update_velocity_field_device;
        blocks = blocks_distr.update_velocity_field_device;
        UpdateVelocityFieldDevice<<<blocks, threads>>>(domain->dev_velocity,
                                                       domain->dev_population,
                                                       domain->dev_density,
                                                       domain->dev_flag_field);
        CUDA_CHECK_ERROR(); 
        
        
        threads = threads_distr.update_population_field_device;
        blocks = blocks_distr.update_population_field_device;
        UpdatePopulationFieldDevice<<<blocks, threads>>>(domain->dev_velocity,
                                                         domain->dev_population,
                                                         domain->dev_density);
        
        CUDA_CHECK_ERROR();

#ifdef DEBUG
        if ((time % parameters.steps_per_report) == 0) {
            int max_index = 0;
            int min_index = 0;
            HANDLE_CUBLAS_ERROR(hipblasIdamax(handle,
                                             parameters.num_lattices,
                                             domain->dev_density,
                                             1,
                                             &max_index));

            HANDLE_CUBLAS_ERROR(hipblasIdamin(handle,
                                             parameters.num_lattices,
                                             domain->dev_density,
                                             1,
                                             &min_index));

            PrintMaxMinDensity<<<1,1>>>(domain->dev_density,
                                        max_index - 1,
                                        min_index - 1,
                                        time);
        }
#endif

#ifdef GRAPHICS
        
        // HACK UPDATE FLAG FIELD
        if (obstacles_added || obstacles_removed) {
            if (obstacles_added) { 
                for (const Point& i: draw_points) {
                    int y = parameters.height - i.y;
                    DrawCircle(i.x, y, WALL, flag_field, parameters);           
                }

                draw_points.clear();
                obstacles_added = false;
            }

            if (obstacles_removed) {
                for (const Point& i: remove_points) {
                    int y = parameters.height - i.y;
                    DrawCircle(i.x, y, FLUID, flag_field, parameters);           
                }

                remove_points.clear();
                obstacles_removed = false;
            }
        
            domain_handler.UpdateFlagField(flag_field, parameters.num_lattices); 

            ScanFlagField(flag_field,
                          domain_handler,
                          bc_handler,
                          parameters,
                          constants,
                          boundary_info);
        }


        ProcessInput(window);

        threads = threads_distr.compute_velocity_magnitude;
        blocks = blocks_distr.compute_velocity_magnitude;
        ComputeVelocityMagnitude<<<blocks, threads>>>(domain->dev_velocity,
                                                          domain->dev_velocity_magnitude);

        hipGraphicsMapResources(1, &resource, NULL);
        hipGraphicsResourceGetMappedPointer((void**)&dev_ptr, &size, resource);

            
        // draw fluid elements
        threads = DEFAULT_THREAD_NUM;
        blocks = ComputeNumBlocks(threads, domain->num_fluid_elements);
        DrawFluid<<<blocks, threads, 0, streams[4]>>>(dev_ptr,
                                                      domain->dev_velocity_magnitude,
                                                      domain->dev_fluid_indices,
                                                      domain->num_fluid_elements);
        CUDA_CHECK_ERROR();
        // draw solid elements
        threads = DEFAULT_THREAD_NUM;
        blocks = ComputeNumBlocks(threads, domain->num_solid_elements);
        DrawObstacles<<<blocks, threads, 0, streams[5]>>>(dev_ptr,
                                                          domain->dev_solid_indices,
                                                          domain->num_solid_elements);
        CUDA_CHECK_ERROR();
            
        SynchStreams<<<1,1>>>();
        // unmap resources to synchronize between rendering and cuda tasks 
        hipGraphicsUnmapResources(1, &resource, NULL);

        glDrawPixels(parameters.width, parameters.height, GL_RGBA, GL_UNSIGNED_BYTE, 0);
        glfwSwapBuffers(window);
        glfwPollEvents(); 
#endif
    
    }
    // END of algorithm
    hipEventRecord(stop, 0);
    
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    float elapsed_time = 0;
    hipEventElapsedTime(&elapsed_time, start, stop); 
    
    double MLUPS = (parameters.num_lattices * parameters.num_time_steps)
                 / (elapsed_time * 1e3);

    printf("MLUPS: %4.6f\n", MLUPS);

    // free HOST recourses
#ifdef GRAPHICS
    hipGraphicsUnregisterResource(resource);
    glfwTerminate();
#endif
    hipblasDestroy(handle);
    free(flag_field);

    // delete streams 
    for (int i = 0; i < NUM_STREAMS; ++i)\
        hipStreamDestroy(streams[i]);
    return 0;
}
