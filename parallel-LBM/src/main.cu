#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <iostream>
#include <stdio.h>
#include <time.h>
#include <iostream>

#include "headers/parameters.h"
#include "headers/init.h"
#include "headers/stub.h"
#include "headers/scanning.h"
#include "headers/boundary_conditions.h"
#include "headers/gnuplot_i.h"
#include "headers/helper.h"
#include "headers/kernels.h"
#include "headers/domain.h"

int main() {

    // choose a proper gpu device with max num threads per block
    hipDeviceProp_t property;
    int gpu_count = 0;
    int gpu_device = 0;
    int max_num_threads_per_block = 0;
    int max_num_registers_per_block = 0;

    HANDLE_ERROR(hipGetDeviceCount(&gpu_count));
    printf("number of gpu devices detected: %d\n", gpu_count);

    for (int gpu_instance = 0; gpu_instance < gpu_count; ++ gpu_instance) {
        HANDLE_ERROR(hipGetDeviceProperties(&property, gpu_instance));
#ifdef DEBUG
        printf(" --- General Information for device %d ---\n", gpu_instance);
        printf("name: %s\n", property.name);
        
        printf("warp size: %d\n", property.warpSize);
        printf("max num. threads per block: %d\n", property.maxThreadsPerBlock);
        printf("max num. registers per block: %d\n", property.regsPerBlock);
        printf("size of constant memory: %d\n", property.totalConstMem);
#endif
        if (property.maxThreadsPerBlock > max_num_threads_per_block) {
            gpu_device = gpu_instance;
            max_num_threads_per_block = property.maxThreadsPerBlock;
            max_num_registers_per_block = property.regsPerBlock;
        }
    }

    // read the enviroment variable "MAX_NUM_THREADS_PER_BLOCK"
    // use the maximum value provide by the DEVICE if the variable has not been defined
    char* str_num_threads = getenv ("MAX_NUM_THREADS_PER_BLOCK");
    if (str_num_threads != NULL) {
        int num_threads = atoi(str_num_threads);
        if (num_threads != 0) {
            max_num_threads_per_block = num_threads;
        }
    } 

    HANDLE_ERROR(hipSetDevice(gpu_device));
    HANDLE_ERROR(hipGetDeviceProperties(&property, gpu_device));
    printf("\n --- %s: device has been chosen --- \n", property.name);
    printf(" --- Number threads per block: %d --- \n", max_num_threads_per_block);
    printf(" --- Number registers per block: %d --- \n", property.regsPerBlock);

    struct SimulationParametes parameters;
    struct BoundaryInfo boundary_info;
    struct Constants constants;

    // read input data
    char parameter_file[] = "parameter.txt";
    char boundary_file[] = "boundary.txt";
    char grid_file[] = "grid.txt";
    ReadInputFilesStub(parameters,
                       boundary_info,
                       constants,
                       parameter_file,
                       boundary_file);
    
    // define cuda grid parameters
    const int MAX_NUM_THREADS = max_num_threads_per_block;
    const int MAX_NUM_BLOCKS = (parameters.num_lattices + MAX_NUM_THREADS) / MAX_NUM_THREADS;

    const int MAX_NUM_USED_REGISTERS_PER_WARP = 35;
    const int MIN_NUM_THREADS = max_num_registers_per_block / MAX_NUM_USED_REGISTERS_PER_WARP;
    const int MIN_NUM_BLOCKS = (parameters.num_lattices + MIN_NUM_THREADS) / MIN_NUM_THREADS;

#ifdef DEBUG
    printf(" --- num elements: %d --- \n", parameters.num_lattices);
    printf(" --- max #threads %d: max #blocks: %d --- \n", MAX_NUM_THREADS, MAX_NUM_BLOCKS);
    printf(" --- min #threads %d: min #blocks: %d --- \n", MIN_NUM_THREADS, MIN_NUM_BLOCKS);

#endif

    // allocate constant data into the DEVICE
    CopyConstantsToDevice(parameters,
                          constants,
                          boundary_info,
                          coords,
                          weights);

#ifdef DEBUG    
    CheckConstMemoryCopy<<<1,1>>>();
    hipDeviceSynchronize();
#endif

    gnuplot_ctrl *velocity_frame;
    gnuplot_ctrl *density_frame;

    velocity_frame = gnuplot_init();
    density_frame = gnuplot_init();

    SetupGnuPlots(velocity_frame, density_frame, parameters);

    // allocate memory in the HOST
    int *flag_field = (int*)calloc(parameters.num_lattices, sizeof(int));
    real *density = (real*)calloc(parameters.num_lattices, sizeof(real));
    real *velocity_magnitude = (real*)calloc(parameters.num_lattices, sizeof(real));

    InitFlagFieldStub(flag_field,
                      grid_file,
                      parameters);
    
    // allocate and init DOMAIN on the DEVICE
    DomainHandler domain_handler;
    domain_handler.InitDomainOnDevice(parameters,
                                      flag_field,
                                      MAX_NUM_THREADS,
                                      MAX_NUM_BLOCKS);

    const Domain *domain = domain_handler.GetDeviceData();

    // allocate and init BOUNDARIES on the DEVICE
    BoundaryConditionsHandler bc_handler;
    
    ScanFlagField(flag_field,
                  bc_handler,
                  parameters,
                  constants,
                  boundary_info);

    const BoundaryConditions *boundaries = bc_handler.GetDeviceData();

    int threads = 0;
    int blocks = 0;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // START of algorighm
    hipEventRecord(start, 0);
    for (int time = 0; time < parameters.num_time_steps; ++time) {
        
        // perform streaming step 
        StreamDevice<<<MAX_NUM_BLOCKS, MAX_NUM_THREADS>>>(domain->dev_population,
                                                          domain->dev_swap_buffer,
                                                          domain->dev_flag_field);
        //CUDA_CHECK_ERROR(); 
        
        // apply boundary consitions
        if (boundaries->num_wall_elements != 0) {
            threads = min(boundaries->num_wall_elements, MAX_NUM_THREADS);
            blocks = (parameters.num_lattices + threads) / threads;
            TreatNonSlipBC<<<blocks, threads>>>(boundaries->bc_wall_indices,
                                                domain->dev_swap_buffer,
                                                boundaries->num_wall_elements); 
            //CUDA_CHECK_ERROR();
        }

        if (boundaries->num_moving_wall_elements != 0) {
            threads = min(boundaries->num_moving_wall_elements, MAX_NUM_THREADS);
            blocks = (parameters.num_lattices + threads) / threads;
            TreatSlipBC<<<blocks, threads>>>(boundaries->bc_moving_wall_indices,
                                         boundaries->bc_moving_wall_data,
                                         domain->dev_density,
                                         domain->dev_swap_buffer,
                                         boundaries->num_moving_wall_elements);
            //CUDA_CHECK_ERROR();
        }

        if (boundaries->num_inflow_elements != 0) {
            threads = min(boundaries->num_inflow_elements, MAX_NUM_THREADS);
            blocks = (parameters.num_lattices + threads) / threads;
            TreatInflowBC<<<blocks, threads>>>(boundaries->bc_inflow_indices,
                                               boundaries->bc_inflow_data,
                                               domain->dev_density,
                                               domain->dev_swap_buffer,
                                               boundaries->num_inflow_elements);
            //CUDA_CHECK_ERROR();
        }

        if (boundaries->num_outflow_elements != 0) {
            threads = min(boundaries->num_outflow_elements, MAX_NUM_THREADS);
            blocks = (parameters.num_lattices + threads) / threads;
            TreatOutflowBC<<<blocks, threads>>>(boundaries->bc_outflow_indices,
                                                domain->dev_velocity,
                                                domain->dev_density,
                                                domain->dev_swap_buffer,
                                                boundaries->num_outflow_elements);
            //CUDA_CHECK_ERROR();
        }


        HANDLE_ERROR(hipDeviceSynchronize());
        domain_handler.SwapPopulationFields(); 
       
        // perform collision step 
        UpdateDensityFieldDevice<<<MAX_NUM_BLOCKS, MAX_NUM_THREADS>>>(domain->dev_density,
                                                                      domain->dev_population,
                                                                      domain->dev_flag_field);

        //CUDA_CHECK_ERROR(); 

        
        UpdateVelocityFieldDevice<<<MAX_NUM_BLOCKS, MAX_NUM_THREADS>>>(domain->dev_velocity,
                                                                       domain->dev_population,
                                                                       domain->dev_density,
                                                                       domain->dev_flag_field);
        //CUDA_CHECK_ERROR(); 

        
        UpdatePopulationFieldDevice<<<MIN_NUM_BLOCKS, MIN_NUM_THREADS>>>(domain->dev_velocity,
                                                                         domain->dev_population,
                                                                         domain->dev_density);
        //CUDA_CHECK_ERROR(); 
        
        /*
        threads = 468;
        blocks = (parameters.num_lattices + threads) / threads; 
        UpdatePopulationFieldDevice<<<blocks, threads>>>(domain->dev_velocity,
                                                         domain->dev_population,
                                                         domain->dev_density);
        */

#ifdef DEBUG



        if ((time % parameters.steps_per_report) == 0) {

            HANDLE_ERROR(hipMemcpy(density,
                                    domain->dev_density,
                                    parameters.num_lattices * sizeof(real),
                                    hipMemcpyDeviceToHost));
         
            real max_density = *std::max_element(density,
                                    density + parameters.num_lattices);
            real min_density = *std::min_element(density,
                                density + parameters.num_lattices);

            std::cout << "time step: " << time << "; ";
            std::cout << "max density: " << max_density << "; ";
            std::cout << "min density "  << min_density << std::endl;
        }
#endif

#ifdef GRAPHICS
        
        if ((time % parameters.steps_per_report) == 0) {
        
            ComputeVelocityMagnitude<<<MAX_NUM_BLOCKS, MAX_NUM_THREADS>>>(domain->dev_velocity,
                                                                          domain->dev_velocity_magnitude);
 

            HANDLE_ERROR(hipMemcpy(velocity_magnitude,
                                    domain->dev_velocity_magnitude,
                                    parameters.num_lattices * sizeof(real),
                                    hipMemcpyDeviceToHost));

            DisplayResults(velocity_magnitude, velocity_frame, parameters);
            // DisplayResults(velocity, velocity_frame,
            //               density, density_frame);
        }
#endif
    }
    // END of algorithm
    hipEventRecord(stop, 0);
    
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    float elapsed_time = 0;
    hipEventElapsedTime(&elapsed_time, start, stop); 
    
    double MLUPS = (parameters.num_lattices * parameters.num_time_steps)
                 / (elapsed_time * 1e3);

    printf("MLUPS: %4.6f\n", MLUPS);

    // free HOST recourses
    gnuplot_close(velocity_frame);
    gnuplot_close(density_frame);

    
    // free DEVICE resources
    free(flag_field);
    free(density);
    free(velocity_magnitude);
    return 0;
}
